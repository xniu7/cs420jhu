/*******************************************************************************
*
*  smooth.cu
*
*  This provides a CUDA implementation of a kernel smooother.
*   http://en.wikipedia.org/wiki/Kernel_smoother
*  The particular smoother in this file is a nearest neighbor smoother
*  in order to keep the code as simple to understand as possible.
*
*  This is implemeneted for 2-d square grids.
*
*  Parameters of note:
*    dataWidth -- size of the data is dataWidth^2
*    halfWidth -- region around point x,y to smooth
*        k smooths box with corners [x-k,y-k] to [x+k,y+k]
*
*  The smoothed region is only defined for the interior that has the kernel
*   defined inside the boundary, e.g. for dataWidth=10, halfWidth=2 the
*   region from 2,2 to 7,7 will be smoothed. 
*
********************************************************************************/

/*******************************************************************************
*
*  CUDA concepts
*
*  This file shows how to use many features of CUDA:
*     2d grids
*     pitch allocation
*     shared memory
*
********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

// Data is of size dataWidth * dataWidth
//const unsigned int dataWidth = 4112;

// Parameter to express the smoothing kernel halfwidth
//const unsigned int halfWidth = 8;

// Size of the CUDA threadBlock
//const unsigned int blockWidth = 16;

/* Small values good for testing */

// Data is of size dataWidth * dataWidth
const unsigned int dataWidth = 14;

// Parameter to express the smoothing kernel halfwidth
const unsigned int halfWidth = 1;
const unsigned int kernelWidth = halfWidth*2+1;

// Size of the CUDA threadBlock
const unsigned int blockWidth = 4;





/*------------------------------------------------------------------------------
* Name: NNSmoothKernel
* Action:  The CUDA kernel that implements kernel smoothing.
*             Yuck, that's two senses of kernel.
*-----------------------------------------------------------------------------*/
__global__ void NNSmoothKernel ( float* pFieldIn, float* pFieldOut, size_t pitch )
{ 
  extern __shared__ float shared[][blockWidth+kernelWidth-1];

  // pitch is in bytes, figure out the number of elements for addressing
  unsigned pitchels = pitch/sizeof(float);

  // compute the halfwidth-1 of the kernel
  unsigned koffset = (kernelWidth-1)/2;


  // Construct the 2d shared memory array it needs to be blockWidth+(kernelWidth-1)/2 square
  // Each node loads one element
  shared[threadIdx.x][threadIdx.y] = 
    pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                   +  blockIdx.x * blockDim.x + threadIdx.x ];

  // And determines if it needs to load it's x-neigbor
  if ( threadIdx.x < kernelWidth -1 )
  {
    shared[threadIdx.x + blockWidth][threadIdx.y] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + blockWidth ];
  }

  // And determines if it needs to load it's y-neigbor
  if ( threadIdx.y < kernelWidth -1 )
  {
    shared[threadIdx.x][threadIdx.y + blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x];
  }

  // And determines if it needs to load it's xy-neigbor
  if ( ( threadIdx.y < kernelWidth -1 ) && ( threadIdx.x < kernelWidth -1 ))
  {
    shared[threadIdx.x + blockWidth][threadIdx.y + blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + blockWidth];
  }

  __syncthreads();

  pFieldOut [ (threadIdx.y+koffset)*pitchels + threadIdx.x+koffset ] = shared [threadIdx.x][threadIdx.y];


  // Variable to accumulate the smoothed value
  float value = 0.0;

  // The grid indexes start from 
  unsigned xindex = ( blockIdx.x * blockDim.x + threadIdx.x) + koffset; 
  unsigned yindex = ( blockIdx.y * blockDim.y + threadIdx.y) + koffset; 

  // Get the value from the kernel
  for ( unsigned j=0; j<kernelWidth; j++ )
  {
    for ( unsigned i=0; i<kernelWidth; i++ )
    {
      value += shared [threadIdx.x+i] [threadIdx.y+j];
    }
  }
  
  // Divide by the number of elements in the kernel
  value /= kernelWidth*kernelWidth;

  // Write the value out 
  pFieldOut [ yindex*pitchels + xindex ] = value;


} 


/*------------------------------------------------------------------------------
* Name:  SmoothField
* Action:  Host entry point to kernel smoother
*-----------------------------------------------------------------------------*/
bool SmoothField ( float* pHostFieldIn, float *pHostFieldOut ) 
{ 
  float * pDeviceFieldIn = 0;
  float * pDeviceFieldOut = 0;

  size_t pitch, pitchout;

  struct timeval ta, tb, tc, td;

  // Check the grid dimensions and extract parameters.  See top description about restrictions
//  assert ((( kernelWidth -1 )%2) == 0 );     // Width is odd
//  assert((gridWidth-(kernelWidth-1) % blockWidth == 0 );

  gettimeofday ( &ta, NULL );

  // Place the data set on device memory
  hipMallocPitch((void**)&pDeviceFieldIn, &pitch, dataWidth*sizeof(float), dataWidth ); 
  hipMemcpy2D ( pDeviceFieldIn, pitch,
                 pHostFieldIn, dataWidth*sizeof(float), dataWidth*sizeof(float), dataWidth,
                 hipMemcpyHostToDevice); 

  // Allocate the output
  hipMallocPitch((void**)&pDeviceFieldOut, &pitchout, dataWidth*sizeof(float), dataWidth ); 

  gettimeofday ( &tb, NULL );

  // Construct a 2d grid/block
  const dim3 DimBlock ( blockWidth, blockWidth );
  const dim3 DimGrid ( (dataWidth-(kernelWidth-1))/blockWidth , 
                       (dataWidth-(kernelWidth-1))/blockWidth );
  const unsigned shmemSize = ( blockWidth + kernelWidth -1 ) * ( blockWidth + kernelWidth -1 ) * sizeof (float);

  // Invoke the kernel
  NNSmoothKernel <<<DimGrid,DimBlock, shmemSize>>> ( pDeviceFieldIn, pDeviceFieldOut, pitch ); 

  gettimeofday ( &tc, NULL );

  // Retrieve the results
  hipMemcpy2D(pHostFieldOut, dataWidth*sizeof(float), 
               pDeviceFieldOut, pitch, dataWidth*sizeof(float), dataWidth,
               hipMemcpyDeviceToHost); 

  gettimeofday ( &td, NULL );


  if ( ta.tv_usec < td.tv_usec )
  {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec, td.tv_usec - ta.tv_usec );
  } else {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec - 1, 1000000 - td.tv_usec + ta.tv_usec );
  }

  if ( tb.tv_usec < tc.tv_usec )
  {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec, tc.tv_usec - tb.tv_usec );
  } else {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec - 1, 1000000 - tc.tv_usec + tb.tv_usec );
  }

  return true;
}



/*------------------------------------------------------------------------------
* Name:  initField
* Action:  Initialize a field to predictable values.
*    This is a useful format for debugging, because values 
*    accumulate to their initial value.
*-----------------------------------------------------------------------------*/
void initField ( unsigned dim, float* pField )
{
  for ( unsigned j=0; j<dim; j++ )
  {
    for ( unsigned i=0; i<dim; i++ )
    {
      pField[j*dim+i] = j + i;
    }
  }
}


/*------------------------------------------------------------------------------
* Name:  main
* Action:  Entry point
*-----------------------------------------------------------------------------*/
int main ()
{
  // Create the input field
  float *field = (float *) malloc ( dataWidth * dataWidth * sizeof(float));
  initField ( dataWidth, field );

  // Create the output field
  float *out = (float *) malloc ( dataWidth * dataWidth * sizeof(float));

  // Call the kernel
  SmoothField ( field, out );

  // Print the output field (for debugging purposes.
  unsigned koffset = (kernelWidth-1)/2;
  for ( unsigned j=0; j< dataWidth; j++ )
  {
    for ( unsigned i=0; i< dataWidth; i++ )
    {
      if ( ( i >= koffset ) && 
           ( j >= koffset ) &&
           ( i < ( dataWidth - koffset )) &&
           ( j < ( dataWidth - koffset )) )
      {
        printf ("%4.4f, ", out[j*dataWidth + i]);
      }
      else
      {
        printf ("%4.4f, ", 0.0f );
      }
    }  
    printf ("\n");
  }

}
